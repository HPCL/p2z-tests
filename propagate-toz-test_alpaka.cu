#include "hip/hip_runtime.h"
/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <alpaka/alpaka.hpp>
#include <functional>
#include <iostream>

#ifndef bsize
#define bsize 16
#endif
#ifndef ntrks
#define ntrks 9600
#endif

#define nb    ntrks/bsize
#define nevts 100
#define smear 0.1

#ifndef NITER
#define NITER 100
#endif

size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}

struct ATRK {
  float par[6];
  float cov[21];
  int q;
  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};

float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0); 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult; 
  call = !call; 
  return (mu + sigma * (float) X1);
}

MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}
//
float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }
//
float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }
//
float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}
float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }
//
void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
void setx    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 0, val); }
void sety    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 1, val); }
void setz    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 2, val); }
void setipt  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 3, val); }
void setphi  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 4, val); }
void settheta(MP6F* bpars, size_t it, float val){ setpar(bpars, it, 5, val); }
//
void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  setpar(&(*btracks).par,it,ipar,val);
}
void setx    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 0, val); }
void sety    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 1, val); }
void setz    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 2, val); }
void setipt  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 3, val); }
void setphi  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 4, val); }
void settheta(MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 5, val); }

const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
//
float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }
//
float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }
//
float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPHIT* bhits = bHit(hits, ev, ib);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }

MPTRK* prepareTracks(ATRK inputtrk) {
  MPTRK* result = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK)); //fixme, align?
  //using DevHost = alpaka::dev::DevCpu;
  //using PltfHost = alpaka::pltf::Pltf<DevHost>;
  //DevHost const devHost(alpaka::pltf::getDevByIdx<PltfHost>(0u));
  //using Data = MPTRK;
  //using Dim = alpaka::dim::DimInt<1u>;
  //using Idx = std::size_t;
  //using BufHost = alpaka::mem::buf::Buf<DevHost,Data,Dim,Idx>;
  //BufHost bufhostA(alpaka::mem::buf::alloc<Data, Idx>(devHost, nevts*nb*sizeof(MPTRK)));
  //Data * result(alpaka::mem::view::getPtrNative(bufhostA));
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
	//par
	for (size_t ip=0;ip<6;++ip) {
	  result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
	}
	//cov
	for (size_t ip=0;ip<21;++ip) {
	  result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip];
	}
	//q
	result[ib + nb*ie].q.data[it] = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//fixme check
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT inputhit) {
  MPHIT* result = (MPHIT*) malloc(nevts*nb*sizeof(MPHIT));  //fixme, align?
  //using DevHost = alpaka::dev::DevCpu;
  //using PltfHost = alpaka::pltf::Pltf<DevHost>;
  //DevHost const devHost(alpaka::pltf::getDevByIdx<PltfHost>(0u));
  //using Data = MPHIT;
  //using Dim = alpaka::dim::DimInt<1u>;
  //using Idx = std::size_t;
  //using BufHost = alpaka::mem::buf::Buf<DevHost,Data,Dim,Idx>;
  //BufHost bufhostA(alpaka::mem::buf::alloc<Data, Idx>(devHost, nevts*nb*sizeof(MPHIT)));
  //Data * result(alpaka::mem::view::getPtrNative(bufhostA));
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
  	//pos
  	for (size_t ip=0;ip<3;++ip) {
  	  result[ib + nb*ie].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
  	}
  	//cov
  	for (size_t ip=0;ip<6;++ip) {
  	  result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
  	}
      }
    }
  }
  printf("result: %f\n",result[0].pos.data[0]);
  return result;
}

#define N bsize
//#pragma acc routine vector nohost
template< typename TAcc>
void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C, TAcc const & acc) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
// #pragma acc loop vector
    using Dim = alpaka::dim::Dim<TAcc>;
    using Idx = alpaka::idx::Idx<TAcc>;
    using Vec = alpaka::vec::Vec<Dim, Idx>;

    Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
    Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
//#pragma omp simd
  for (int n = threadIdx[1]; n < N; n+=threadExtent[1])
  //for (int n = 0; n < N; ++n)
  {
    c[ 0*N+n] = b[ 0*N+n] + a[ 2*N+n]*b[ 3*N+n] + a[ 3*N+n]*b[ 6*N+n] + a[ 4*N+n]*b[10*N+n] + a[ 5*N+n]*b[15*N+n];
    c[ 1*N+n] = b[ 1*N+n] + a[ 2*N+n]*b[ 4*N+n] + a[ 3*N+n]*b[ 7*N+n] + a[ 4*N+n]*b[11*N+n] + a[ 5*N+n]*b[16*N+n];
    c[ 2*N+n] = b[ 3*N+n] + a[ 2*N+n]*b[ 5*N+n] + a[ 3*N+n]*b[ 8*N+n] + a[ 4*N+n]*b[12*N+n] + a[ 5*N+n]*b[17*N+n];
    c[ 3*N+n] = b[ 6*N+n] + a[ 2*N+n]*b[ 8*N+n] + a[ 3*N+n]*b[ 9*N+n] + a[ 4*N+n]*b[13*N+n] + a[ 5*N+n]*b[18*N+n];
    c[ 4*N+n] = b[10*N+n] + a[ 2*N+n]*b[12*N+n] + a[ 3*N+n]*b[13*N+n] + a[ 4*N+n]*b[14*N+n] + a[ 5*N+n]*b[19*N+n];
    c[ 5*N+n] = b[15*N+n] + a[ 2*N+n]*b[17*N+n] + a[ 3*N+n]*b[18*N+n] + a[ 4*N+n]*b[19*N+n] + a[ 5*N+n]*b[20*N+n];
    c[ 6*N+n] = b[ 1*N+n] + a[ 8*N+n]*b[ 3*N+n] + a[ 9*N+n]*b[ 6*N+n] + a[10*N+n]*b[10*N+n] + a[11*N+n]*b[15*N+n];
    c[ 7*N+n] = b[ 2*N+n] + a[ 8*N+n]*b[ 4*N+n] + a[ 9*N+n]*b[ 7*N+n] + a[10*N+n]*b[11*N+n] + a[11*N+n]*b[16*N+n];
    c[ 8*N+n] = b[ 4*N+n] + a[ 8*N+n]*b[ 5*N+n] + a[ 9*N+n]*b[ 8*N+n] + a[10*N+n]*b[12*N+n] + a[11*N+n]*b[17*N+n];
    c[ 9*N+n] = b[ 7*N+n] + a[ 8*N+n]*b[ 8*N+n] + a[ 9*N+n]*b[ 9*N+n] + a[10*N+n]*b[13*N+n] + a[11*N+n]*b[18*N+n];
    c[10*N+n] = b[11*N+n] + a[ 8*N+n]*b[12*N+n] + a[ 9*N+n]*b[13*N+n] + a[10*N+n]*b[14*N+n] + a[11*N+n]*b[19*N+n];
    c[11*N+n] = b[16*N+n] + a[ 8*N+n]*b[17*N+n] + a[ 9*N+n]*b[18*N+n] + a[10*N+n]*b[19*N+n] + a[11*N+n]*b[20*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = 0;
    c[14*N+n] = 0;
    c[15*N+n] = 0;
    c[16*N+n] = 0;
    c[17*N+n] = 0;
    c[18*N+n] = b[ 6*N+n];
    c[19*N+n] = b[ 7*N+n];
    c[20*N+n] = b[ 8*N+n];
    c[21*N+n] = b[ 9*N+n];
    c[22*N+n] = b[13*N+n];
    c[23*N+n] = b[18*N+n];
    c[24*N+n] = a[26*N+n]*b[ 3*N+n] + a[27*N+n]*b[ 6*N+n] + b[10*N+n] + a[29*N+n]*b[15*N+n];
    c[25*N+n] = a[26*N+n]*b[ 4*N+n] + a[27*N+n]*b[ 7*N+n] + b[11*N+n] + a[29*N+n]*b[16*N+n];
    c[26*N+n] = a[26*N+n]*b[ 5*N+n] + a[27*N+n]*b[ 8*N+n] + b[12*N+n] + a[29*N+n]*b[17*N+n];
    c[27*N+n] = a[26*N+n]*b[ 8*N+n] + a[27*N+n]*b[ 9*N+n] + b[13*N+n] + a[29*N+n]*b[18*N+n];
    c[28*N+n] = a[26*N+n]*b[12*N+n] + a[27*N+n]*b[13*N+n] + b[14*N+n] + a[29*N+n]*b[19*N+n];
    c[29*N+n] = a[26*N+n]*b[17*N+n] + a[27*N+n]*b[18*N+n] + b[19*N+n] + a[29*N+n]*b[20*N+n];
    c[30*N+n] = b[15*N+n];
    c[31*N+n] = b[16*N+n];
    c[32*N+n] = b[17*N+n];
    c[33*N+n] = b[18*N+n];
    c[34*N+n] = b[19*N+n];
    c[35*N+n] = b[20*N+n];
  }
}

//#pragma acc routine vector nohost
template< typename TAcc>
void MultHelixPropTranspEndcap(const MP6x6F* A, const MP6x6F* B, MP6x6SF* C, TAcc const & acc) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
// #pragma acc loop vector
    using Dim = alpaka::dim::Dim<TAcc>;
    using Idx = alpaka::idx::Idx<TAcc>;
    using Vec = alpaka::vec::Vec<Dim, Idx>;

    Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
    Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
//#pragma omp simd
  for (int n = threadIdx[1]; n < N; n+=threadExtent[1])
  //for (int n = 0; n < N; ++n)
  {
    c[ 0*N+n] = b[ 0*N+n] + b[ 2*N+n]*a[ 2*N+n] + b[ 3*N+n]*a[ 3*N+n] + b[ 4*N+n]*a[ 4*N+n] + b[ 5*N+n]*a[ 5*N+n];
    c[ 1*N+n] = b[ 6*N+n] + b[ 8*N+n]*a[ 2*N+n] + b[ 9*N+n]*a[ 3*N+n] + b[10*N+n]*a[ 4*N+n] + b[11*N+n]*a[ 5*N+n];
    c[ 2*N+n] = b[ 7*N+n] + b[ 8*N+n]*a[ 8*N+n] + b[ 9*N+n]*a[ 9*N+n] + b[10*N+n]*a[10*N+n] + b[11*N+n]*a[11*N+n];
    c[ 3*N+n] = b[12*N+n] + b[14*N+n]*a[ 2*N+n] + b[15*N+n]*a[ 3*N+n] + b[16*N+n]*a[ 4*N+n] + b[17*N+n]*a[ 5*N+n];
    c[ 4*N+n] = b[13*N+n] + b[14*N+n]*a[ 8*N+n] + b[15*N+n]*a[ 9*N+n] + b[16*N+n]*a[10*N+n] + b[17*N+n]*a[11*N+n];
    c[ 5*N+n] = 0;
    c[ 6*N+n] = b[18*N+n] + b[20*N+n]*a[ 2*N+n] + b[21*N+n]*a[ 3*N+n] + b[22*N+n]*a[ 4*N+n] + b[23*N+n]*a[ 5*N+n];
    c[ 7*N+n] = b[19*N+n] + b[20*N+n]*a[ 8*N+n] + b[21*N+n]*a[ 9*N+n] + b[22*N+n]*a[10*N+n] + b[23*N+n]*a[11*N+n];
    c[ 8*N+n] = 0;
    c[ 9*N+n] = b[21*N+n];
    c[10*N+n] = b[24*N+n] + b[26*N+n]*a[ 2*N+n] + b[27*N+n]*a[ 3*N+n] + b[28*N+n]*a[ 4*N+n] + b[29*N+n]*a[ 5*N+n];
    c[11*N+n] = b[25*N+n] + b[26*N+n]*a[ 8*N+n] + b[27*N+n]*a[ 9*N+n] + b[28*N+n]*a[10*N+n] + b[29*N+n]*a[11*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = b[27*N+n];
    c[14*N+n] = b[26*N+n]*a[26*N+n] + b[27*N+n]*a[27*N+n] + b[28*N+n] + b[29*N+n]*a[29*N+n];
    c[15*N+n] = b[30*N+n] + b[32*N+n]*a[ 2*N+n] + b[33*N+n]*a[ 3*N+n] + b[34*N+n]*a[ 4*N+n] + b[35*N+n]*a[ 5*N+n];
    c[16*N+n] = b[31*N+n] + b[32*N+n]*a[ 8*N+n] + b[33*N+n]*a[ 9*N+n] + b[34*N+n]*a[10*N+n] + b[35*N+n]*a[11*N+n];
    c[17*N+n] = 0;
    c[18*N+n] = b[33*N+n];
    c[19*N+n] = b[32*N+n]*a[26*N+n] + b[33*N+n]*a[27*N+n] + b[34*N+n] + b[35*N+n]*a[29*N+n];
    c[20*N+n] = b[35*N+n];
  }
}

//#pragma acc routine vector nohost
template< typename TAcc>
void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar,
//void ALPAKA_FN_ACC propagateToZ(TAcc const & acc, const MP6x6SF* inErr, const MP6F* inPar,
		  const MP1I* inChg, const MP3F* msP,
	                MP6x6SF* outErr, MP6F* outPar,
 		struct MP6x6F* errorProp, struct MP6x6F* temp, TAcc const & acc) {
    using Dim = alpaka::dim::Dim<TAcc>;
    using Idx = alpaka::idx::Idx<TAcc>;
    using Vec = alpaka::vec::Vec<Dim, Idx>;

    Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
    Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  //
//    using Dim = alpaka::dim::Dim<TAcc>;
//    using Idx = alpaka::idx::Idx<TAcc>;
//    using Vec = alpaka::vec::Vec<Dim, Idx>;
//    using Vec1 = alpaka::vec::Vec<alpaka::dim::DimInt<1u>, Idx>;
//
//    Vec const globalThreadIdx    = alpaka::idx::getIdx<alpaka::Grid, alpaka::Threads>(acc);
//    Vec const globalThreadExtent = alpaka::workdiv::getWorkDiv<alpaka::Grid, alpaka::Threads>(acc);
// #pragma acc loop vector
  for (size_t it=threadIdx[1];it<bsize;it+=threadExtent[1]) {	
  //for (size_t it=0;it<bsize;it++) {	
    const float zout = z(msP,it);
    //printf ("running prop: %f\n",zout);
    const float k = q(inChg,it)*100/3.8;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1./ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float alpha = deltaZ*sinT*ipt(inPar,it)/(cosT*k);
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    setx(outPar,it, x(inPar,it) + k*(pxin*sina - pyin*(1.-cosa)) );
    sety(outPar,it, y(inPar,it) + k*(pyin*sina + pxin*(1.-cosa)) );
    setz(outPar,it,zout);
    setipt(outPar,it, ipt(inPar,it));
    setphi(outPar,it, phi(inPar,it)+alpha );
    settheta(outPar,it, theta(inPar,it) );
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
    
    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)/cosT;
    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*ipt(inPar,it))-k*(cosP*sina-sinP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k/ipt(inPar,it))*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*cosT);
    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)/cosT;
    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*ipt(inPar,it))-k*(sinP*sina+cosP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k/ipt(inPar,it))*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*cosT);
    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT/(cosT*k);
    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ/(cosT*k);
    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ/(cosT*cosT*k);
  }
  //
  MultHelixPropEndcap(errorProp, inErr, temp,acc);
  MultHelixPropTranspEndcap(errorProp, temp, outErr,acc);
  //MultHelixPropEndcap(errorProp, inErr, temp);
  //MultHelixPropTranspEndcap(errorProp, temp, outErr);
}





template< typename TAcc>
void ALPAKA_FN_ACC alpaka_kernel(TAcc const & acc, MPTRK* trk, MPHIT* hit, MPTRK* outtrk){
    //printf ("running kernel\n");
    using Dim = alpaka::dim::Dim<TAcc>;
    using Idx = alpaka::idx::Idx<TAcc>;
    using Vec = alpaka::vec::Vec<Dim, Idx>;

    //Vec const globalThreadIdx    = alpaka::idx::getIdx<alpaka::Grid, alpaka::Threads>(acc);
    //Vec const globalThreadExtent = alpaka::workdiv::getWorkDiv<alpaka::Grid, alpaka::Threads>(acc);
//
  // for (size_t ie=globalThreadIdx[2];ie<nevts;ie+=globalThreadExtent[2]) { // loop over events
    // for (size_t ib=globalThreadIdx[1];ib<nb;ib+=globalThreadExtent[1]) { // loop over bunches of tracks
    Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
    Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
    Vec const blockIdx    = alpaka::idx::getIdx<alpaka::Grid, alpaka::Blocks>(acc);
    Vec const blockExtent = alpaka::workdiv::getWorkDiv<alpaka::Grid, alpaka::Blocks>(acc);

   for (size_t ie=blockIdx[0];ie<nevts;ie+=blockExtent[0]) { // loop over events
     for (size_t ib=threadIdx[0];ib<nb;ib+=threadExtent[0]) { // loop over bunches of tracks
     //for (size_t ib=blockIdx[0];ib<nb;ib+=blockExtent[0]) { // loop over bunches of tracks
   //for (size_t ie=0;ie<nevts;++ie) { // loop over events
     //for (size_t ib=0;ib<nb;++ib) { // loop over bunches of tracks
       //
       //printf ("running kernel: %f\n",trk[0].par.data[0]);
       const MPTRK* btracks = bTk(trk, ie, ib);
       const MPHIT* bhits = bHit(hit, ie, ib);
       MPTRK* obtracks = bTk(outtrk, ie, ib);
 	     struct MP6x6F errorProp, temp;
       //printf ("running kernel: %f\n",(btracks->par).data[0]);
       //
       propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, &(*obtracks).cov, &(*obtracks).par,
	   &errorProp, &temp, acc); // vectorized function
    }
  }
}
















int main (int argc, char* argv[]) {

  using Dim = alpaka::dim::DimInt<2>;
  using Idx = std::size_t;
  // set type of accelerator
  //using Acc = alpaka::acc::AccCpuSerial<Dim, Idx>;
  //using Acc = alpaka::acc::AccCpuOmp4<Dim, Idx>;
  //using Acc = alpaka::acc::AccCpuThreads<Dim, Idx>;
  //using Acc = alpaka::acc::AccCpuOmp2Threads<Dim, Idx>;
  //using Acc = alpaka::acc::AccCpuOmp2Blocks<Dim, Idx>;
  /////////////
  //using Acc = alpaka::acc::AccCpuTbbBlocks<Dim, Idx>;
  using Acc = alpaka::acc::AccGpuCudaRt<Dim, Idx>;

  using DevAcc = alpaka::dev::Dev<Acc>;
  using PltfAcc = alpaka::pltf::Pltf<DevAcc>;

  using QueueProperty = alpaka::queue::Blocking;
  using QueueAcc = alpaka::queue::Queue<Acc,QueueProperty>;

  // select device
  DevAcc const devAcc(alpaka::pltf::getDevByIdx<PltfAcc>(0u));

  //make queue on device
  QueueAcc queue(devAcc);


  using Vec = alpaka::vec::Vec<Dim,Idx>;
  //Vec const elementsPerThread(Vec::all(static_cast<Idx>(4)));
  //Vec const threadsPerBlock(Vec::all(static_cast<Idx>(8)));
  //Vec const blocksPerGrid(static_cast<Idx>(4),static_cast<Idx>(1));//,static_cast<Idx>(2));
  //static constexpr uint64_t blockSize = alpaka::dim::DimInt<2>::value; 
  //Idx blockCount = static_cast<Idx>(alpaka::acc::getAccDevProps<Acc,DevAcc>(devAcc).m_multiProcessorCount*8);

  Vec const elementsPerThread(Vec::all(static_cast<Idx>(1)));
  Vec const threadsPerBlock(Vec::all(static_cast<Idx>(8)));
  //Vec const threadsPerBlock(Vec::all(static_cast<Idx>(8)));
  Vec const blocksPerGrid(static_cast<Idx>(4),static_cast<Idx>(1));//,static_cast<Idx>(2));

  using WorkDiv = alpaka::workdiv::WorkDivMembers<Dim, Idx>;
  //WorkDiv const workDiv( static_cast<Idx>(blockCount), static_cast<Idx>(blockSize),block);
  //WorkDiv workDiv{ static_cast<Idx>(blockCount), static_cast<Idx>(blockSize),static_cast<Idx>(1)};
  //WorkDiv const workDiv( blocksPerGrid, static_cast<Idx>(blockSize),elementsPerThread);
  WorkDiv const workDiv( blocksPerGrid, threadsPerBlock,elementsPerThread);



   int itr;
   ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1,
     {1, 0, 17, 16, 36, 35, 33, 34, 59, 58, 70, 85, 101, 102, 116, 117, 132, 133, 152, 169, 187, 202}
   };

   AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
   };

   printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
   printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
	                                       inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
	                                       inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
   printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);
   
   printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
   printf("NITER=%d\n", NITER);
   
   long start, end, setup_start, setup_end;
   long start2, end2;
   struct timeval timecheck;

   gettimeofday(&timecheck, NULL);
   setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
   MPTRK* trk = prepareTracks(inputtrk);
   MPHIT* hit = prepareHits(inputhit);
   printf("host: %f\n",hit[0].pos.data[0]);
   gettimeofday(&timecheck, NULL);
   setup_end = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   printf("done preparing!\n");
   
   MPTRK* outtrk = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK));
  
  //using Data_hit = MPHIT;
  //using Data_trk = MPTRK;
  //using Dim = alpaka::dim::DimInt<1>;
  //using Idx = std::size_t;
  //using BufHost_hit = alpaka::mem::buf::Buf<DevAcc,Data_hit,Dim,Idx>;
  //using BufHost_trk = alpaka::mem::buf::Buf<DevAcc,Data_trk,Dim,Idx>;
  //BufHost_hit bufhit_dev(alpaka::mem::buf::alloc<Data_hit, Idx>(devAcc, nevts*nb*sizeof(MPHIT)));
  //BufHost_trk buftrk_dev(alpaka::mem::buf::alloc<Data_trk, Idx>(devAcc, nevts*nb*sizeof(MPTRK)));
  //BufHost_trk bufouttrk_dev(alpaka::mem::buf::alloc<Data_trk, Idx>(devAcc, nevts*nb*sizeof(MPTRK)));
  //using DevHost = alpaka::dev::DevCpu;
  //using PltfHost = alpaka::pltf::Pltf<DevHost>;
  //DevHost const devHost(alpaka::pltf::getDevByIdx<PltfHost>(0u));
  //BufHost_trk bufouttrk(alpaka::mem::buf::alloc<Data_trk, Idx>(devHost, nevts*nb*sizeof(MPTRK)));
  //Data_trk * outtrk_dev(alpaka::mem::view::getPtrNative(bufouttrk_dev));
  //Data_trk * outtrk(alpaka::mem::view::getPtrNative(bufouttrk));
  //Data_trk * trk_dev(alpaka::mem::view::getPtrNative(buftrk_dev));
  //Data_hit * hit_dev(alpaka::mem::view::getPtrNative(bufhit_dev));



   // for (size_t ie=0;ie<nevts;++ie) {
   //   for (size_t it=0;it<ntrks;++it) {
   //     printf("ie=%lu it=%lu\n",ie,it);
   //     printf("hx=%f\n",x(&hit,ie,it));
   //     printf("hy=%f\n",y(&hit,ie,it));
   //     printf("hz=%f\n",z(&hit,ie,it));
   //     printf("tx=%f\n",x(&trk,ie,it));
   //     printf("ty=%f\n",y(&trk,ie,it));
   //     printf("tz=%f\n",z(&trk,ie,it));
   //   }
   // }
  

   printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
   printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
   printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));

   gettimeofday(&timecheck, NULL);
   start2 = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

  //copy host to acc
  printf("test 1\n");
  //alpaka::mem::view::copy(queue,trk_dev,trk,nevts*nb*sizeof(MPTRK));
  printf("test 2\n");
  //alpaka::mem::view::copy(queue,trk_dev->par,trk->par,sizeof(MP6F));
  printf("test 3\n");
  



  //alpaka::mem::view::copy(queue,hit_dev,hit,nevts*nb*sizeof(MPHIT));


   gettimeofday(&timecheck, NULL);
   start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
   for(itr=0; itr<NITER; itr++) {
     alpaka::kernel::exec<Acc>( queue,workDiv,
     [] ALPAKA_FN_ACC (Acc const & acc, MPTRK* trk, MPHIT* hit, MPTRK* outtrk){
     alpaka_kernel(acc, trk,hit,outtrk);
     }, trk, hit, outtrk);

     alpaka::wait::wait(queue);
//   for (size_t ie=0;ie<nevts;++ie) { // loop over events
//     for (size_t ib=0;ib<nb;++ib) { // loop over bunches of tracks
//       //
//       const MPTRK* btracks = bTk(trk, ie, ib);
//       const MPHIT* bhits = bHit(hit, ie, ib);
//       MPTRK* obtracks = bTk(outtrk, ie, ib);
// 	     struct MP6x6F errorProp, temp;
//       //
//       propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, &(*obtracks).cov, &(*obtracks).par,
//	   &errorProp, &temp); // vectorized function
//    }
//  }
  } //end of itr loop
   gettimeofday(&timecheck, NULL);
   end = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
//}

   gettimeofday(&timecheck, NULL);
   end2 = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   // for (size_t ie=0;ie<nevts;++ie) {
   //   for (size_t it=0;it<ntrks;++it) {
   //     printf("ie=%lu it=%lu\n",ie,it);
   //     printf("tx=%f\n",x(&outtrk,ie,it));
   //     printf("ty=%f\n",y(&outtrk,ie,it));
   //     printf("tz=%f\n",z(&outtrk,ie,it));
   //   }
   // }
   
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks, (end-start)*0.001, (end-start)*0.001/(nevts*ntrks));
   printf("data region time=%f (s)\n", (end2-start2)*0.001);
   printf("memory transter time=%f (s)\n", ((end2-start2) - (end-start))*0.001);
   printf("setup time time=%f (s)\n", (setup_end-setup_start)*0.001);
   printf("formatted %i %f %e %f %f %f 0\n",nevts*ntrks, (end-start)*0.001, (end-start)*0.001/(nevts*ntrks), (end2-start2)*0.001,  ((end2-start2) - (end-start))*0.001, (setup_end-setup_start)*0.001);

   float avgx = 0, avgy = 0, avgz = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       avgx += x_;
       avgy += y_;
       avgz += z_;
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0;
   float stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);

//   free(trk);
//   free(hit);
//   free(outtrk);

   return 0;
}
